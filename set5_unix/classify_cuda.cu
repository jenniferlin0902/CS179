#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include "classify_cuda.cuh"
#include "math.h"

/*
 * Arguments:
 * data: Memory that contains both the review LSA coefficients and the labels.
 *       Format decided by implementation of classify.
 * batch_size: Size of mini-batch, how many elements to process at once
 * step_size: Step size for gradient descent. Tune this as needed. 1.0 is sane
 *            default.
 * weights: Pointer to weights vector of length REVIEW_DIM.
 * errors: Pointer to a single float used to describe the error for the batch.
 *         An output variable for the kernel. The kernel can either write the
 *         value of loss function over the batch or the misclassification rate
 *         in the batch to errors.
 */


__global__
void trainLogRegKernel(
    float *data,
    int batch_size,
    int step_size,
	float *weights,
    float *errors)
{

    __shared__ float grad[REVIEW_DIM];
    //float* grad = shared;
    float temp;
    float* err_count = &temp;
    // float* err_count = (float*)(grad + REVIEW_DIM);
    int x_dim = REVIEW_DIM + 1;
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;


    if (threadId < REVIEW_DIM){
        grad[threadId] = 0.0;
        //printf("data = %f\n", data[threadId*x_dim + 50]);
    }
    if (threadId == 0){
        *err_count = 0;
        printf("err_cout = %f\n", *err_count);
    }
    __syncthreads();
    while (threadId < batch_size){
        float y = data[threadId * x_dim + 50];

        for (int i = 0; i < REVIEW_DIM; i++){
            float x = data[threadId * x_dim + i];
            //printf("x = %f\n", x);
            float sub_grad =  x * y / (1 + exp(y * weights[threadId] * x));
            atomicAdd(&(grad[i]), sub_grad);

        }
        threadId += blockDim.x * gridDim.x;
        printf("grad = %f\n", grad[1]);
    }
    __syncthreads();
    threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId == 0){
        for (int i = 0; i < REVIEW_DIM; i++){
           // weights[i] -= step_size * grad[i]/batch_size;
        }
    }

/*
    while (threadId < batch_size){
        float est_y = 0;
        for (int i = 0; i < REVIEW_DIM; i++){
            est_y +=  data[threadId * x_dim + i]* weights[i];
        }
        printf("est_y = %f\n", est_y);
        if (est_y * data[threadId * x_dim + 50] <= 0 ){
            atomicAdd(err_count, 1);
            //printf("here\n");
        }
        threadId += blockDim.x * gridDim.x;
    }
*/
    __syncthreads();
    threadId = blockIdx.x * blockDim.x + threadIdx.x;
    //if (threadId == 0){
        //int temp = *err_count/batch_size;
        *errors = 1.0;

}

/*
 * All parameters have the same meaning as in docstring for trainLogRegKernel.
 * Notably, cudaClassify returns a float that quantifies the error in the
 * minibatch. This error should go down as more training occurs.
 */
float cudaClassify(
    float *data,
    int batch_size, 
    float step_size,
    float *weights, 
    hipStream_t stream)
{
    int block_size = (batch_size < 1024) ? batch_size : 1024;

    // grid_size = CEIL(batch_size / block_size)
    int grid_size = (batch_size + block_size - 1) / block_size;
    int shmem_bytes = 0;



    float *d_errors;
    hipMalloc(&d_errors, sizeof(float));
    hipMemset(d_errors, 0, sizeof(float));

    trainLogRegKernel<<<grid_size, block_size, shmem_bytes, stream>>>(
        data,
        batch_size,
        step_size,
        weights,
        d_errors);

    float h_errors = -1.0;
    hipMemcpy(&h_errors, d_errors, sizeof(float), hipMemcpyDefault);
    hipFree(d_errors);
    return h_errors;
}
