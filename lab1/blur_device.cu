#include "hip/hip_runtime.h"
/* 
 * CUDA blur
 * Kevin Yuh, 2014 
 * Revised by Nailen Matschke, 2016
 */

#include <cstdio>

#include <hip/hip_runtime.h>

#include "blur_device.cuh"


__global__
void cudaBlurKernel(const float *raw_data, const float *blur_v, float *out_data,
    int n_frames, int blur_v_size) {

    // TODO: Fill in the implementation for the GPU-accelerated convolution. 
    //
    // It may be helpful to use the information in the lecture slides, as well
    // as the CPU implementation, as a reference.
    uint thread_index = blockIdx.x * blockDim.x* threadIdx.x;
    int j = 0;
    if (thread_index < n_frames) {
        while(j < blur_v_size) {
            out_data[thread_index] = raw_data[thread_index - j] * blur_v[j];
            j++;
        }
    }
}


void cudaCallBlurKernel(const unsigned int blocks,
        const unsigned int threadsPerBlock,
        const float *raw_data,
        const float *blur_v,
        float *out_data,
        const unsigned int n_frames,
        const unsigned int blur_v_size) {
        
    memset(out_data,0x0, n_frames* sizeof(float));
    cudaBlurKernel<<<<blocks, threadsPerBlock>>>>(raw_data,blur_v, out_data
    n_frames, blur_v_size);
}
